#include "hip/hip_runtime.h"
#include "DocDupDetectorGPU.h"

#include <cstring>
#include <iostream>
#include <algorithm>
#include <cassert>
#include <ctime>
#include <queue>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../Common/Common.h"
#include "../Common/CudaMemoryManager.cuh"

using namespace std;


#define MAX_DUP_DOCUMENTS 10000
#define MAX_HASH_STR_LEN 128

#define MAX_DUP_HASHSTRING_LENGTH (MAX_HASH_STR_LEN + 2)   //7k

#define ROLLING_WINDOW 7
#define THRESHOLD 0.7
#define MINIMUM_SCORE 80
#define BASE 130
#define ALPHA_NUMBER 94

#define MAX_BLOCKS 64       //ÿ��ѭ��������ĵ���
#define MAX_THREADS_PER_BLOCK 64
#define MAX_THREADS (MAX_BLOCKS * MAX_THREADS_PER_BLOCK)

#define INF 0x6fff

typedef short EditDistT;
typedef unsigned int hash_type;


static vector <char*> contents_buffer;
static vector <char*> hashstrs_buffer;
static vector <int>   contents_length;
static vector <int>   hashstrs_length;
//������original data

static vector<int> candies[MAX_DUP_DOCUMENTS];
static vector<int> real_dups[MAX_DUP_DOCUMENTS];
//�����Ǵ�

static vector<pair<int, int> > order_by_length;

static double average_len;
void DocDupDetectorGPU::initialize() {
	contents_buffer.clear();
	hashstrs_buffer.clear();
	average_len = 0.0;
	for(int i=0;i<MAX_DUP_DOCUMENTS;i++) {
		candies[i].clear();
		real_dups[i].clear();
	}
}

void DocDupDetectorGPU::add_document(string doc) {
	//printf("doc length: %d, %s\n", doc.length(), doc.c_str());
	hash_type Tmp = 1;
	for(int i=0;i<ROLLING_WINDOW;i++) Tmp *= BASE;

	char *p = new char[doc.length() + 1];
	strcpy(p, doc.c_str());

	string code = "";
	int block_size = 4;
	while(code=="" || code.length()>MAX_HASH_STR_LEN) {
		code = "";
		for(int i=0, j;i<(int)doc.length();i++) {
			unsigned roll_h = 0, h = 0;
			for(j=i;j<(int)doc.length();j++) {
				h = h * BASE + doc[j];
				roll_h = roll_h * BASE + doc[j];
				if(j-i >= ROLLING_WINDOW) {
					roll_h -= Tmp * doc[j - ROLLING_WINDOW];
				}
				if(roll_h % block_size == block_size - 1) {
					break;
				}
			}
			code += (char)('!' + h % ALPHA_NUMBER);
			i = j;
		}
		block_size *= 2;
	}
	//cout<<block_size<<endl;

	char *h = new char[code.length()+1];
	strcpy(h, code.c_str());

	//cout<<code<<endl;
	printf("doc_id = %d, hash_value = %s\n", (int)contents_buffer.size(), code.c_str());
	average_len += code.length();

	contents_buffer.push_back(p);
	contents_length.push_back(strlen(p));
	hashstrs_buffer.push_back(h);
	hashstrs_length.push_back(strlen(h));
}

//����[b1, b2)�ڵ����д������������ظ����, һ�������tot_blocks = b2 - b1
__global__ void calcDupsByGpu(char **d_hashstrs, int *d_hashstrs_length, int *d_startId, int *d_endedId, int b1, int b2, int doc_num, int *ans_buffer, int *ans_len) {
	int tot_blocks = gridDim.x;
	//assert(blockDim.x == 1);
	int bid_intotal = blockIdx.x;
	
	if(b1 + bid_intotal >= b2) 
		return;

	ans_len[bid_intotal] = 0;
	int start = d_startId[b1 + bid_intotal];
	int ended = d_endedId[b1 + bid_intotal];

	__shared__ EditDistT edit_dis[2][MAX_DUP_HASHSTRING_LENGTH];
	__shared__ char str1[MAX_DUP_HASHSTRING_LENGTH];
	__shared__ char str2[MAX_DUP_HASHSTRING_LENGTH];
	int len1 = d_hashstrs_length[b1+bid_intotal], len2;
	//hipMemcpy(str1+1, d_hashstrs[b1+bid_intotal], sizeof(char) * len1, hipMemcpyDeviceToDevice);
	for(int i=0;i<len1;i++) {
		str1[i+1] = d_hashstrs[b1+bid_intotal][i];
	}
	for(int to = start;to <= ended;to ++) {
		len2 = d_hashstrs_length[to];
		//hipMemcpy(str2+1, d_hashstrs[to], sizeof(char) * len2, hipMemcpyDeviceToDevice);
		for(int i=0;i<len2;i++) {
			str2[i+1] = d_hashstrs[to][i];
		}
		int now = 0;
		for(int j=0;j<=len2;j++) {
			edit_dis[now][j] = j;
		}
		for(int i=1;i<=len1;i++) {
			for(int j=0;j<=len2;j++) {
				edit_dis[!now][j] = INF;
			}
			for(int j=1;j<=len2;j++) {
				edit_dis[!now][j] = (edit_dis[now][j] < edit_dis[!now][j-1] ? edit_dis[now][j] : edit_dis[!now][j-1]) + 1;
				if(edit_dis[now][j-1]+1 < edit_dis[!now][j])
					edit_dis[!now][j] = edit_dis[now][j-1] + 1;
				if(str1[i] == str2[j] && edit_dis[now][j-1] < edit_dis[!now][j])
					edit_dis[!now][j] = edit_dis[now][j-1];
			}
			now = !now;
		}
		//printf("edit-dis[%d - %d] = %d\n", b1 + bid_intotal, to, edit_dis[now][len2]);
		if(edit_dis[now][len2] < len1 * (1.0 - THRESHOLD) &&
			edit_dis[now][len2] < len2 * (1.0 - THRESHOLD)) {
				ans_buffer[bid_intotal * doc_num + ans_len[bid_intotal]++] = to;
				//printf("debug: %d - %d\n", b1 + bid_intotal, to);
		}
	}
}

template <typename T> __device__ T gmin(T a, T b) {
	return a < b ? a : b;
}

#define checkmin(a, b) if((a)>(b))a=b


template <typename T> __device__ T gmax(T a, T b) {
	return a > b ? a : b;
}

//����[b1, b2)�ڵ����д������������ظ����, һ�������tot_blocks = b2 - b1
__global__ void calcDupsByGpu2(char **d_hashstrs, int *d_hashstrs_length, int *d_startId, int *d_endedId, int b1, int b2, int doc_num, int *ans_buffer, int *ans_len) {
	int tot_blocks = gridDim.x;
	//assert(blockDim.x == 1);
	int bid_intotal = blockIdx.x;

	if(b1 + bid_intotal >= b2) 
		return;

	ans_len[bid_intotal] = 0;
	int start = d_startId[b1 + bid_intotal];
	int ended = d_endedId[b1 + bid_intotal];

	__shared__ EditDistT edit_dis[2][MAX_DUP_HASHSTRING_LENGTH];
	__shared__ char str1[MAX_DUP_HASHSTRING_LENGTH];
	__shared__ char str2[MAX_DUP_HASHSTRING_LENGTH];
	int len1 = d_hashstrs_length[b1+bid_intotal], len2;
	//hipMemcpy(str1+1, d_hashstrs[b1+bid_intotal], sizeof(char) * len1, hipMemcpyDeviceToDevice);
	for(int i=0;i<len1;i++) {
		str1[i+1] = d_hashstrs[b1+bid_intotal][i];
	}
	for(int to = start;to <= ended;to ++) {
		len2 = d_hashstrs_length[to];
		//hipMemcpy(str2+1, d_hashstrs[to], sizeof(char) * len2, hipMemcpyDeviceToDevice);
		for(int i=0;i<len2;i++) {
			str2[i+1] = d_hashstrs[to][i];
		}
		int W = gmin(len1, len2) * THRESHOLD + 1;
		int now = 0;
		for(int j=0;j<=len2;j++) {
			edit_dis[now][j] = j;
		}
		bool suc;
		for(int i=1;i<=len1;i++) {
			int sl = gmax(0, i-1-W);
			int sr = gmin(len2, i-1+W);
			int tl = gmax(1, i-W);
			int tr = gmin(len2, i+W);
			suc = false;
			for(int j=tl;j<=tr;j++) {
// 				edit_dis[!now][j] = INF;
// 				if(j>=sl && j<=sr) 
// 					checkmin(edit_dis[!now][j], edit_dis[now][j]+1);
// 				if(j-1>=tl && j-1<=tr)
// 					checkmin(edit_dis[!now][j], edit_dis[!now][j-1]+1);
// //				edit_dis[!now][j] = (edit_dis[now][j] < edit_dis[!now][j-1] ? edit_dis[now][j] : edit_dis[!now][j-1]) + 1;
// 				if(j-1>=sl && j-1<=sr)
// 					checkmin(edit_dis[!now][j], edit_dis[now][j-1]+1);
// // 				if(edit_dis[now][j-1]+1 < edit_dis[!now][j])
// // 					edit_dis[!now][j] = edit_dis[now][j-1] + 1;
// 				
// 				if(str1[i] == str2[j])
// 					checkmin(edit_dis[!now][j], edit_dis[now][j-1]);
// // 				if(str1[i] == str2[j] && edit_dis[now][j-1] < edit_dis[!now][j])
// // 					edit_dis[!now][j] = edit_dis[now][j-1];
// 				if(edit_dis[!now][j] < INF)
// 					suc = true;
 			}
			now = !now;
			if(!suc) break;
		}
		//printf("edit-dis[%d - %d] = %d\n", b1 + bid_intotal, to, edit_dis[now][len2]);
		if(suc && edit_dis[now][len2] < len1 * (1.0 - THRESHOLD) &&
			edit_dis[now][len2] < len2 * (1.0 - THRESHOLD)) {
				ans_buffer[bid_intotal * doc_num + ans_len[bid_intotal]++] = to;
				//printf("debug: %d - %d\n", b1 + bid_intotal, to);
		}
	}
}

//һ��block�������һ��str������str�ı༭���롣
__global__ void calcDupsByGpu3(char **d_hashstrs, int *d_hashstrs_length, int *d_startId, int *d_endedId, int b1, int b2, int doc_num, char *char_map, EditDistT *edit_dis) {
	int totBlocks = gridDim.x;
	int nThreadPerBlock = blockDim.x;
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;

	if(b1 + blockId >= b2) return;

	int tid_intotal = blockId * nThreadPerBlock + threadId;
	EditDistT* dp[2];
	dp[0] = edit_dis + tid_intotal * 2 * MAX_DUP_HASHSTRING_LENGTH;
	dp[1] = dp[0] + MAX_DUP_HASHSTRING_LENGTH;

	__shared__ char str2[MAX_DUP_HASHSTRING_LENGTH];
	int len2 = d_hashstrs_length[b1+blockId], len1;
	if(threadId == 0) {
		for(int i=0;i<len2;i++) {
			str2[i+1] = d_hashstrs[b1+blockId][i];
		}
	}
	int start = d_startId[b1 + blockId];
	int ended = d_endedId[b1 + blockId];
	
	for(int ite = 0;;ite++) {
		int to = start + ite * nThreadPerBlock + threadId;
		if(to > ended) break;
		len1 = d_hashstrs_length[to];
		//hipMemcpy(str1+1, d_hashstrs[to], sizeof(char) * len1, hipMemcpyDeviceToDevice);
		char *str1 = d_hashstrs[to];

		int now = 0;
		for(int j=0;j<=len2;j++) {
			dp[now][j] = j;
		}
		for(int i=1;i<=len1;i++) {
			char ch = str1[i-1];
			for(int j=0;j<=len2;j++) {
				dp[!now][j] = INF;
			}
			EditDistT a, b, c, d;
			for(int j=1;j<=len2;j++) {
				b = dp[now][j];
				c = dp[!now][j-1];
				d = dp[now][j-1];
				a = (b < c ? b : c) + 1;
				if(d+1 < a)
					a = d + 1;
				if(str2[j] == ch && d < a)
					a = d;
				dp[!now][j] = a;
			}
			now = !now;
		}
		//printf("edit-dis[%d - %d] = %d\n", b1 + bid_intotal, to, dp[now][len1]);
		if(dp[now][len2] < len2 * (1.0 - THRESHOLD) &&
			dp[now][len2] < len1 * (1.0 - THRESHOLD)) {
				char_map[blockId * MAX_DUP_DOCUMENTS + to] = 'y';
				//ans_buffer[bid_intotal * doc_num + ans_len[bid_intotal]++] = to;
				//printf("debug: %d - %d\n", b1 + bid_intotal, to);
		}
	}
}

static char *  h_contents[MAX_DUP_DOCUMENTS];
static int     h_contents_length[MAX_DUP_DOCUMENTS]; //����

static char *  h_hashstrs[MAX_DUP_DOCUMENTS];
static int     h_hashstrs_length[MAX_DUP_DOCUMENTS]; //��ϣ��
//�����Ǿ���order_by_lengthת��֮���data, ֵ����GPU�еĵ�ַ

static int h_ans_len[MAX_BLOCKS];
static int h_ans[MAX_DUP_DOCUMENTS];

//��������ʱ����

static int startId[MAX_DUP_DOCUMENTS];
static int endedId[MAX_DUP_DOCUMENTS];

static CudaMemoryManager<char> memo_mana_c;
static CudaMemoryManager<int>  memo_mana_i;
static CudaMemoryManager<EditDistT> memo_mana_s;


void DocDupDetectorGPU::useMethod1(int doc_num, char **d_hashstrs, int *d_hashstrs_length, int *d_startId, int *d_endedId) {
	clock_t ttt = clock();
	int *d_ans_buf = memo_mana_i.gpu_malloc(MAX_BLOCKS * doc_num); 
	int *d_ans_len = memo_mana_i.gpu_malloc(MAX_BLOCKS);
	safeCudaCall(hipMemset(d_ans_len, 0, sizeof(int) * MAX_BLOCKS));

	for(int b1=0;b1<doc_num;b1+=MAX_BLOCKS) {
		int b2 = min(b1 + MAX_BLOCKS, doc_num);
		LOG(logger, "Processing docs[%6d, %6d)......", b1, b2);
		int t = clock();
		calcDupsByGpu<<<MAX_BLOCKS, 1>>>(d_hashstrs, d_hashstrs_length, d_startId, d_endedId, b1, b2, doc_num, d_ans_buf, d_ans_len);
		hipDeviceSynchronize();
		printf("time used: %lf s\n", (clock()-t) / (double)CLOCKS_PER_SEC);

		t = clock();
		safeCudaCall(hipMemcpy(h_ans_len, d_ans_len, sizeof(int) * MAX_BLOCKS, hipMemcpyDeviceToHost));
		for(int i=0;i<b2-b1;i++) if(h_ans_len[i] > 0) {
			safeCudaCall(hipMemcpy(h_ans, d_ans_buf + i * doc_num, h_ans_len[i] * sizeof(int), hipMemcpyDeviceToHost));
			//printf("h_ans_len[%d] = %d, id1 = %d\n", b1+i, h_ans_len[i], order_by_length[b1+i].second);
			for(int j=0;j<h_ans_len[i];j++) {
				int id1 = order_by_length[b1+i].second;
				int id2 = order_by_length[h_ans[j]].second;
				if(id1 == id2) continue;
				candies[id1].push_back(id2);
				if(id1 > id2) candies[id2].push_back(id1);
			}
		}
		printf("data copy and insert: %lf s\n", (clock()-t) / (double)CLOCKS_PER_SEC);
	}

	printf("calculateDups time: %lf s\n", (clock()-ttt) / (double)CLOCKS_PER_SEC);
}

void DocDupDetectorGPU::useMethod3(int doc_num, char **d_hashstrs, int *d_hashstrs_length, int *d_startId, int *d_endedId) {
	clock_t ttt = clock();
	EditDistT *edit_dis = memo_mana_s.gpu_malloc(MAX_THREADS * 2 * MAX_DUP_HASHSTRING_LENGTH);
	char *char_map = memo_mana_c.gpu_malloc(MAX_BLOCKS * MAX_DUP_DOCUMENTS);
	char *h_char_map = (char*)malloc(MAX_BLOCKS * MAX_DUP_DOCUMENTS); //freed
	for(int b1=0;b1<doc_num;b1+=MAX_BLOCKS) {
		int b2 = min(b1 + MAX_BLOCKS, doc_num);
		printf("Processing docs[%6d, %6d)......", b1, b2);
		int t = clock();
		safeCudaCall(hipMemset(char_map, 0, sizeof(char)*(MAX_BLOCKS * MAX_DUP_DOCUMENTS)));
		calcDupsByGpu3<<<MAX_BLOCKS, MAX_THREADS_PER_BLOCK>>>(d_hashstrs, d_hashstrs_length, d_startId, d_endedId, b1, b2, doc_num, char_map, edit_dis);
		hipDeviceSynchronize();
		LOG(logger, "time used: %lf s", (clock()-t) / (double)CLOCKS_PER_SEC);

		t = clock();
		safeCudaCall(hipMemcpy(h_char_map, char_map, sizeof(char) * MAX_BLOCKS * MAX_DUP_DOCUMENTS, hipMemcpyDeviceToHost));
		for(int i=0;i<b2-b1;i++) {
			for(int j=0;j<MAX_DUP_DOCUMENTS;j++) if(h_char_map[i*MAX_DUP_DOCUMENTS + j] == 'y') {
				int id1 = order_by_length[b1+i].second;
				int id2 = order_by_length[j].second;
				if(id1 == id2) continue;
				candies[id1].push_back(id2);
				if(id1 > id2) candies[id2].push_back(id1);
			}
		}
		LOG(logger, "data copy and insert: %lf s", (clock()-t) / (double)CLOCKS_PER_SEC);
	}
	free(h_char_map);
	LOG(logger, "calculateDups time: %lf s", (clock()-ttt) / (double)CLOCKS_PER_SEC);
}

void DocDupDetectorGPU::calculate_dups() {
	LOG(logger, "%s", "Begin calculate doc dups by GPU.");
	LOG(logger, "average_len = %lf", average_len / contents_buffer.size());
	int ttt = clock();
	int doc_num = contents_buffer.size();
	order_by_length.clear();
	double sumL = 0.0, maxL = 0.0;
	for(int i=0;i<doc_num;i++) {
		order_by_length.push_back(make_pair(hashstrs_length[i], i));
		sumL += hashstrs_length[i];
		maxL = max(maxL, (double)hashstrs_length[i]);
	}
	sort(order_by_length.begin(), order_by_length.end());

	//copy the document content into gpu memory.
	int maxLD = 0;
// 	for(int i=0;i<doc_num;i++) {
// 		int id = order_by_length[i].second;
// 		char *d_c = memo_mana_c.gpu_malloc(contents_length[id]);
// 		safeCudaCall(hipMemcpy(d_c, contents_buffer[id], sizeof(char)*contents_length[id], hipMemcpyHostToDevice));
// 		h_contents[i] = d_c;
// 		h_contents_length[i] = contents_length[id];
// 		maxLD = max(maxLD, h_contents_length[i]);
// 	}
// 	char **d_contents = memo_mana_c.copy_bucks(h_contents, doc_num);


	for(int i=0;i<doc_num;i++) {
		int id = order_by_length[i].second;
		char *d_c = memo_mana_c.gpu_malloc(hashstrs_length[id]);
		//LOG(logger, "%d[old %d]th hashstrs: %s", i, id, hashstrs_buffer[id]); 
		safeCudaCall(hipMemcpy(d_c, hashstrs_buffer[id], sizeof(char)*hashstrs_length[id], hipMemcpyHostToDevice));
		h_hashstrs[i] = d_c;
		h_hashstrs_length[i] = hashstrs_length[id];
	}
	char **d_hashstrs = memo_mana_c.copy_bucks(h_hashstrs, doc_num);

	int *d_hashstrs_length = memo_mana_i.gpu_malloc(doc_num);
	safeCudaCall(hipMemcpy(d_hashstrs_length, h_hashstrs_length, doc_num * sizeof(int), hipMemcpyHostToDevice));

	for(int i=0, x=0, y=0;i<doc_num;i++) {
		int xl = order_by_length[i].first * THRESHOLD;
		int yl = order_by_length[i].first * (2.0-THRESHOLD);
		while(x<doc_num && order_by_length[x].first < xl) x++;
		while(y+1<doc_num && order_by_length[y+1].first <= yl) y++;
		startId[i] = x;
		endedId[i] = min(y, i);
		//startId[i] = 0;
		//endedId[i] = doc_num-1;
		LOG(logger, "Test range of doc_%06d is[%6d, %6d]", i, x, y);
	
	}

	int *d_startId = memo_mana_i.gpu_malloc(doc_num);
	safeCudaCall(hipMemcpy(d_startId, startId, sizeof(int) * doc_num, hipMemcpyHostToDevice));
	int *d_endedId = memo_mana_i.gpu_malloc(doc_num);
	safeCudaCall(hipMemcpy(d_endedId, endedId, sizeof(int) * doc_num, hipMemcpyHostToDevice));

	for(int i=0;i<doc_num;i++) {
		candies[i].clear();
	}

	//useMethod1(doc_num, d_hashstrs, d_hashstrs_length, d_startId, d_endedId);
	useMethod3(doc_num, d_hashstrs, d_hashstrs_length, d_startId, d_endedId);

	core_time = clock() - ttt;
	LOG(logger, "hashstrs average length: %lf, max length: %lf", sumL / doc_num, maxL);
	LOG(logger, "content max length: %d", maxLD);
	LOG(logger, "calculateDups total time: %lf s", (clock()-ttt) / (double)CLOCKS_PER_SEC);
}

vector<int> DocDupDetectorGPU::get_candidate_dup_docs(int did) {
	return candies[did];
}

void DocDupDetectorGPU::refine(){
	for(int id1 = 0; id1<(int)hashstrs_buffer.size();id1++){
		const char* doc1=contents_buffer[id1];
		vector<int> item = candies[id1];
		for(int i=0;i<item.size();i++){
			const char *doc2=contents_buffer[item[i]];
			if(id1 < item[i] && score(doc1, doc2) >= MINIMUM_SCORE){
				real_dups[id1].push_back(item[i]);
				real_dups[item[i]].push_back(id1);
			}
		}
		LOG(logger, "refine doc_id = %d ended, candidates = %d, real_dups = %d", id1, (int)item.size(), real_dups[id1].size());
		if(item.size() && item.size() < 5) {
			for(int i=0;i<(int)item.size();i++) cout<<item[i]<<' '; cout<<endl;
		}
	}
}

vector<int> DocDupDetectorGPU::get_real_dup_docs(int did) {
	return real_dups[did];
}

DocDupDetectorGPU::~DocDupDetectorGPU() {
	for(int i=0;i<(int)contents_buffer.size();i++) {
		delete[] contents_buffer[i];
		delete[] hashstrs_buffer[i];
	}
}
